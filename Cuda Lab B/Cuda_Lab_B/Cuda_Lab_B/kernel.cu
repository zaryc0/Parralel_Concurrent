﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
    printf("Thread ID : %d Block: ID: %d", threadIdx.x, blockIdx.x);
}

int main()
{
    const int size = 5;
    const int a[size] = { 1, 2, 3, 4, 5 };
    const int b[size] = { 10, 20, 30, 40, 50 };
    int c[size] = { 0 };

    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    
    // Add vectors in parallel.
    hipError_t cudaStatus;


    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_c, c, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //addKernel << <1, 5 >> > (dev_c, dev_a, dev_b);
    //addKernel << <2, 3 >> > (dev_c, dev_a, dev_b);
    //addKernel << <2, 4 >> > (dev_c, dev_a, dev_b);
    //addKernel << <2, 5 >> > (dev_c, dev_a, dev_b);
    //addKernel << <2, 6 >> > (dev_c, dev_a, dev_b);
    //addKernel << <3, 2 >> > (dev_c, dev_a, dev_b);
    //addKernel << <3, 3 >> > (dev_c, dev_a, dev_b);

    hipDeviceSynchronize();
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n", c[0], c[1], c[2], c[3], c[4]);


Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return 0;
}

