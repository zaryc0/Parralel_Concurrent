#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void addKernel(int c[][32], const int a[][32], const int b[][32])
{
    int i = threadIdx.x;
    int j = threadIdx.y;

    c[i][j] = a[i][j] + b[i][j];
    //printf("Thread ID : (%d,%d) Block: ID: (%d,%d)\n", threadIdx.x, threadIdx.y, blockIdx.x,blockIdx.y);
}

int main()
{
    const int size = 32;
    int a[size][size];
    int b[size][size];
    int c[size][size];

    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            a[i][j] = i + j;
            b[i][j] = (i + j) * 10;
        }
    }
    
    int( *dev_a)[size];
    int( *dev_b)[size];
    int( *dev_c)[size];

    // Add vectors in parallel.
    hipError_t cudaStatus;


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_c, c, size * size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    addKernel << <1, dim3(32,32) >> > (dev_c, dev_a, dev_b);

    hipDeviceSynchronize();
    hipMemcpy(c, dev_c, size * size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    printf("a + b =...\n");
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            printf("c[%d,%d] = %d \n", i, j, c[i][j]);
        }
    }


Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return 0;
}

