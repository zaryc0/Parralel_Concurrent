#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_vector_types.h>

 // includes, cuda
#include <hip/hip_runtime_api.h>
#include <iostream>
#include "hitable.h"
#include "hitable_list.h"
#include "sphere.h"
#include "vec3.h"
#include "ray.h"

typedef unsigned int uint;
typedef unsigned char uchar;


hipArray* d_imageArray = 0;
__device__ static int ticks = 1;
__device__ static int colour_index = 1;
__device__ static vec3 sphere_centres[3];
__device__ static vec3 sphere_velocitys[3];
__device__ static vec3 sphere_colours[3];

__device__ vec3 castRay(const ray& r, hitable** world) {
    hit_record rec;

    if ((*world)->hit(r, 0.0, FLT_MAX, rec)) {
        vec3 result = 0.5f * vec3(rec.normal.x() + 1.0f , rec.normal.y() + 1.0f, rec.normal.z() + 1.0f);
        return result * (rec.colour/255);
    }
    else {
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5f * (unit_direction.y() + 1.0f);
        return (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
    }
}
__global__ void create_world(hitable** d_list, hitable** d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) 
    {
        //define wall vec3s
        vec3 lef = vec3(-10002.0,        0,        0);
        vec3 rig = vec3( 10002.0,        0,        0);
        vec3 bot = vec3(       0, -10002.0,        0);
        vec3 top = vec3(       0,  10002.0,        0);
        vec3 bac = vec3(       0,        0, -10001.0);

        //define the colours
        vec3 blank = vec3(255, 255, 255);
        int number_of_colours = 6;
        vec3 colour_list[6];
        colour_list[0] = vec3(066, 245, 242);
        colour_list[1] = vec3(245, 066, 224);
        colour_list[2] = vec3(245, 242, 066);
        colour_list[3] = vec3(000, 255, 000);
        colour_list[4] = vec3(000, 000, 255);
        colour_list[5] = vec3(255, 000, 000);

        //initialise sphers statics
        if (sphere_velocitys[0].x() == 0 && sphere_velocitys[0].y() == 0 && sphere_velocitys[0].z() == 0) { sphere_velocitys[0] = vec3(0.01, 0.05, 0); sphere_colours[0] = vec3(150, 150, 150); }
        if (sphere_velocitys[1].x() == 0 && sphere_velocitys[1].y() == 0 && sphere_velocitys[1].z() == 0) { sphere_velocitys[1] = vec3(-0.03, -0.01, 0); sphere_colours[1] = vec3(150, 150, 150); }
        if (sphere_velocitys[2].x() == 0 && sphere_velocitys[2].y() == 0 && sphere_velocitys[2].z() == 0) { sphere_velocitys[2] = vec3(0.05, -0.02, 0); sphere_colours[0] = vec3(150, 150, 150);
        }

        int wall_size = 10000;
        //Create the walls
        *(d_list + 0) = new sphere(lef, blank, vec3(0, 0, 0), wall_size);
        *(d_list + 1) = new sphere(rig, blank, vec3(0, 0, 0), wall_size);
        *(d_list + 2) = new sphere(bot, blank, vec3(0, 0, 0), wall_size);
        *(d_list + 3) = new sphere(top, blank, vec3(0, 0, 0), wall_size);
        *(d_list + 4) = new sphere(bac, blank, vec3(0, 0, 0), wall_size);
        //Modify balls
        for (int i = 0; i < 3; i++)
        {
            //move balls
            sphere_centres[i] += sphere_velocitys[i];
            //Test Collisions
            if (sphere_centres[i].x() - 0.2 <= lef.x() + wall_size || sphere_centres[i].x() + 0.2 >= rig.x() - wall_size)
            {
                sphere_colours[i] = colour_list[colour_index];
                colour_index++;
                if (colour_index > 5)
                {
                    colour_index = 0;
                }
                sphere_velocitys[i] = vec3(sphere_velocitys[i].x() * -1, sphere_velocitys[i].y(), sphere_velocitys[i].z());
            }
            if (sphere_centres[i].y() - 0.2 <= bot.y() + wall_size || sphere_centres[i].y() + 0.2 >= top.y() - wall_size)
            {
                sphere_colours[i] = colour_list[colour_index];
                colour_index++;
                if (colour_index > 5)
                {
                    colour_index = 0;
                }
                sphere_velocitys[i] = vec3(sphere_velocitys[i].x(), sphere_velocitys[i].y() * -1, sphere_velocitys[i].z());
            }
            *(d_list + i + 5) = new sphere(sphere_centres[i], sphere_colours[i], sphere_velocitys[i], 0.2);
        }
        *d_world = new hitable_list(d_list, 8);
    }
}
__global__ void free_world(hitable** d_list, hitable** d_world) {
    delete* (d_list + 0);
    delete* (d_list + 1);
    delete* (d_list + 2);
    delete* (d_list + 3);
    delete* (d_list + 4);
    delete* (d_list + 5);
    delete* d_world;
}

__global__ void d_render(uchar4* d_output, uint width, uint height, hitable** d_world)
{
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;
    uint i = y * width + x;

    float u = x / (float)width;
    float v = y / (float)height;

    u = 2.0 * u - 1.0;
    v = -(2.0 * v - 1.0);
    //scale u by aspect ratio
    u *= width / (float)height;

    u *= 2.0;
    v *= 2.0;

    vec3 eye = vec3(0, 0.5, 1.5);
    float distFromEyeToImg = 1.0;
    if ((x < width) && (y < height))
    {
        vec3 pixelPos = vec3(u, v, eye.z() - distFromEyeToImg);
        ray r;
        r.O = eye;
        r.Dir = pixelPos - eye;

        vec3 col = castRay(r, d_world);
        float red = col.x();
        float green = col.y();
        float blue = col.z();
        d_output[i] = make_uchar4(red * 255, green * 255, blue * 255, 0);
    }
}

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

extern "C" void freeTexture() {

    checkCudaErrors(hipFreeArray(d_imageArray));
}

// render image using CUDA
extern "C" 
    void render(int width, int height,  dim3 blockSize, dim3 gridSize, uchar4 * output) 
{
    hitable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, 8 * sizeof(hitable*)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hitable*)));
    create_world << <1, 1 >> > (d_list, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    d_render << <gridSize, blockSize >> > (output, width, height, d_world);
    getLastCudaError("kernel failed");
}
#endif