#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_vector_types.h>

 // includes, cuda
#include <hip/hip_runtime_api.h>

typedef unsigned int uint;
typedef unsigned char uchar;


hipArray* d_imageArray = 0;


__global__ void d_render(uchar4* d_output, uint width, uint height) {
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    uint i = __umul24(y, width) + x;
    float u = x / (float)width;
    float v = y / (float)height;
    u = 2.0 * u - 1.0;
    v = -(2.0 * v - 1.0);

    uint c = 255;
    float r = 0.5;
    //c= ((((x & 0x80) == 0) ^ ((y & 0x80)) == 0));
    if ((x < width) && (y < height)) 
    {
        float dist = sqrtf(powf(u - (0) ,2) + powf(v - (0),2));

        if(dist<r)
        {
            d_output[i] = make_uchar4(0x00 , 0x00 , 0xff , 0);
        }
        else
        {
            d_output[i] = make_uchar4(0x66, 0x99, 0x00, 0);
        }
    }
}


extern "C" void freeTexture() {

    checkCudaErrors(hipFreeArray(d_imageArray));
}

// render image using CUDA
extern "C" void render(int width, int height,  dim3 blockSize, dim3 gridSize,
     uchar4 * output) {


            d_render << <gridSize, blockSize >> > (output, width, height);


    getLastCudaError("kernel failed");
}

#endif